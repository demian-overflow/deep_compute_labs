
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello() {
    printf("Hello, World from GPU thread %d!\n", threadIdx.x);
}

int main() {
    printf("Hello from CPU!\n");
    
    // Launch kernel with 5 threads
    cuda_hello<<<1, 5>>>();
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    return 0;
} 